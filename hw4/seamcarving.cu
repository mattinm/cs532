#include "hip/hip_runtime.h"
/* W B Langdon at MUN 10 May 2007
 * Program to demonstarte use of OpenGL's glDrawPixels
 */

#ifdef _WIN32
#include <windows.h>
#endif

#ifdef __APPLE__
#  include <OpenGL/gl.h>
#  include <OpenGL/glu.h>
#  include <GLUT/glut.h>
#else
#  include <GL/gl.h>
#  include <GL/glu.h>
#  include <GL/glut.h>
#endif

#include <string>
#include <queue>
#include <iostream>
#include <iomanip>
#include <sstream>
#include "math.h"

#include "lodepng.h"

#define INDEX4(x, y, z, width) ((((y) * (width) + (x)) * 4) + (z))
#define INDEX(x, y, width) ((y) * (width) + (x))

using std::cerr;
using std::cout;
using std::endl;
using std::string;
using std::ostream;
using std::setw;
using std::right;
using std::left;
using std::fixed;
using std::vector;
using std::priority_queue;
using std::setprecision;

unsigned int window_width, window_height;
unsigned int window_size;

/**
 * the pixels are now in the vector "image", 4 bytes per pixel, ordered RGBARGBA..., use it as texture, draw it, ...
 */

std::vector<unsigned char> image; //the raw pixels
unsigned char *inverted_image;
int *dirs;
float *greyscale;
float *vals;
float *costs;
int *seam;

// cuda variables
#define TILE_SIZE   512
int *gpu_dirs;
float *gpu_costs;
float *gpu_vals;
unsigned char *gpu_inverted_image;

hipError_t err;
dim3 dimBlock(TILE_SIZE, 1, 1);
dim3 dimGrid(1,1,1);

int seams_to_remove;
long start_time;

#define CUDAASSERT(x) \
    if ((x) != hipSuccess) { \
        cout << hipGetErrorString(x) << " in file " << __FILE__ << " at line " << __LINE__ << endl; \
        exit(EXIT_FAILURE); \
    }

//find the pixel, and the RGBA (z is 0, 1, 2, 3)  part of that pixel
static int POSITION4(int x, int y, int z) {
    return (((y * window_width) + x) * 4) + z;
}

static int POSITION(int x, int y) {
    return ((y * window_width) + x);
}

__global__ void gpu_calc_costs(float *costs, float *vals, int *dirs, int width, int height)
{
    // tiled storage for the local
    extern __shared__ float shared_costs[];
    float *current_costs = &shared_costs[width];

    // determine our location
    int x;
    int index;

    // fill in the top row first
    for (x = threadIdx.x; x < width; x += TILE_SIZE) {
        index = INDEX(x, height-1, width);

        // initialize our top row 
        shared_costs[x] = vals[index];
        costs[index] = shared_costs[x];
    }

    // go by column
    float cost_left, cost_up, cost_right, cost;
    for (int y = height - 2; y >= 0; --y) {
        // sync before starting
        __syncthreads();

        for (x = threadIdx.x; x < width; x += TILE_SIZE) {
            index = INDEX(x, y, width);

            // the left edges must know the last right value
            if (x == 0) {
                cost_left = 100000.0f;
            } else {
                cost_left = shared_costs[x-1];
            }

            cost_up = shared_costs[x];

            // the right edges must know the next left value
            if (x == (width-1)) {
                cost_right = 100000.0f;
            } else {
                cost_right = shared_costs[x+1];
            }

            // update our current shared cost and direction
            cost = vals[index];
            if (cost_left < cost_up && cost_left < cost_right) {
                cost += cost_left;
                dirs[index] = -1;
            } else if (cost_right < cost_left && cost_right < cost_up) {
                cost += cost_right;
                dirs[index] = 1;
            } else {
                cost += cost_up;
                dirs[index] = 0;
            }

            // update our main memory
            costs[index] = cost;
            current_costs[x] = cost;
        }

        // sync before flipping the shared_costs array to next
        __syncthreads();

        // update the shared_costs
        for (x = threadIdx.x; x < width; x += TILE_SIZE) {
            shared_costs[x] = current_costs[x];
        }
    }
}

/**
 *  The display function gets called repeatedly, updating the visualization of the simulation
 */
int count = 0;
void display() {
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    if (count < seams_to_remove) {
        gpu_calc_costs<<<dimGrid, dimBlock, (window_width*2)>>>(gpu_costs, gpu_vals, gpu_dirs, window_width, window_height);
        err = hipGetLastError();
        CUDAASSERT(err);

        err = hipMemcpy(costs, gpu_costs, sizeof(*costs) * window_size, hipMemcpyDeviceToHost);
        CUDAASSERT(err);

        err = hipMemcpy(dirs, gpu_dirs, sizeof(*dirs) * window_size, hipMemcpyDeviceToHost);
        CUDAASSERT(err);

        //calculate the same to remove
        //first get the min cost at the bottom row
        float min_val = 50000;
        for (int x = 0; x < (window_width - count); x++) {
            if (costs[POSITION(x,0)] < min_val) {
                min_val = costs[POSITION(x,0)];
                seam[0] = x;
    //            cout << "min_val now " << min_val << " for x: " << x << endl;
            }
        }

        for (int y = 1; y < window_height; y++) {
    //        cout << "calculating seam[" << y << "]: based on seam[" << (y-1) << "]: " << seam[y-1] << " + " << dirs[POSITION(seam[y-1], y-1)] << endl;
            seam[y] = seam[y-1] + dirs[POSITION(seam[y-1], y-1)];
        }

        /*
        cout << "seam:";
        for (int i = 0; i < window_height; i++) {
            cout << " " << seam[i];
        }
        cout << endl << endl;
        */


        //remove one line with the seamcarving algorithm
        for (int y = 0; y < window_height; y++) {
            int x;
            for (x = seam[y]; x < (window_width - count) - 1; x++) {
                inverted_image[POSITION4(x, y, 0)] = inverted_image[POSITION4(x+1, y, 0)];
                inverted_image[POSITION4(x, y, 1)] = inverted_image[POSITION4(x+1, y, 1)];
                inverted_image[POSITION4(x, y, 2)] = inverted_image[POSITION4(x+1, y, 2)];
                inverted_image[POSITION4(x, y, 3)] = inverted_image[POSITION4(x+1, y, 3)];

                vals[POSITION(x, y)] = vals[POSITION(x+1, y)];            
            }

            inverted_image[POSITION4(x, y, 0)] = 0;
            inverted_image[POSITION4(x, y, 1)] = 0;
            inverted_image[POSITION4(x, y, 2)] = 0;
            inverted_image[POSITION4(x, y, 3)] = 0;

            vals[POSITION(x, y)] = 0;
        }

    } else if (count == seams_to_remove) {
        cout << "It took " << (time(NULL) - start_time) << " seconds to remove " << seams_to_remove << " seams." << endl;
    }

    count++;

    glDrawPixels(window_width, window_height, GL_RGBA, GL_UNSIGNED_BYTE, inverted_image);

    glFlush();
    glutSwapBuffers();

    glutPostRedisplay();
}

int main(int argc, char** argv) {
    if (argc != 3) {
        cerr << "Invalid arguments." << endl;
        cerr << "Proper usage:" << endl;
        cerr << "\t" << argv[0] << " <png filename> <seams to remove>" << endl;
        exit(1);
    }

    const char* filename = argv[1];
    seams_to_remove = atoi(argv[2]);
    start_time = time(NULL);

    //decode
    unsigned error = lodepng::decode(image, window_width, window_height, filename);

    //if there's an error, display it
    if(error) std::cout << "decoder error " << error << ": " << lodepng_error_text(error) << std::endl;

    //the pixels are now in the vector "image", 4 bytes per pixel, ordered RGBARGBA..., use it as texture, draw it, ...

    window_size = window_width * window_height;
    inverted_image = new unsigned char[window_size * 4];

    dirs = new int[window_size];
    costs = new float[window_size];
    greyscale = new float[window_size];
    vals = new float[window_size];
    seam = new int[window_height];

    //the PNG is inverted in height as to how the pixels are displayed, so we need to flip it.
    for (int y = 0; y < window_height; y++) {
        for (int x = 0; x < window_width; x++) {
            inverted_image[POSITION4(x, (window_height - 1 - y), 0)] = image[POSITION4(x, y, 0)];
            inverted_image[POSITION4(x, (window_height - 1 - y), 1)] = image[POSITION4(x, y, 1)];
            inverted_image[POSITION4(x, (window_height - 1 - y), 2)] = image[POSITION4(x, y, 2)];
            inverted_image[POSITION4(x, (window_height - 1 - y), 3)] = image[POSITION4(x, y, 3)];
        }
    }

    //get the average of all the color channels and use that as the value for each pixel.
    for (int y = 0; y < window_height; y++) {
        for (int x = 0; x < window_width; x++) {
            greyscale[POSITION(x, y)] = (inverted_image[POSITION4(x, y, 0)] +
                                         inverted_image[POSITION4(x, y, 1)] +
                                         inverted_image[POSITION4(x, y, 2)] +
                                         inverted_image[POSITION4(x, y, 3)]) * 0.25;
        }
    }

    //Calculate the gradient for every pixel
    for (int y = 0; y < window_height; y++) {
        for (int x = 0; x < window_width; x++) {
            float result = 0;

            if (x > 0)                  result += fabs(greyscale[POSITION(x, y)] - greyscale[POSITION(x-1, y)]);
            if (x < window_width - 1)   result += fabs(greyscale[POSITION(x, y)] - greyscale[POSITION(x+1, y)]);
            if (y > 0)                  result += fabs(greyscale[POSITION(x, y)] - greyscale[POSITION(x, y-1)]);
            if (y < window_height - 1)  result += fabs(greyscale[POSITION(x, y)] - greyscale[POSITION(x, y+1)]);

            vals[POSITION(x, y)] = result;
//            cout << "vals[" << x << ", " << y << "]: " << vals[POSITION(x,y)] << endl;
        }
    }

    // done with greyscale
    delete[] greyscale;
    greyscale = NULL;

    // setup our gpu memory
    err = hipMalloc((void **) &gpu_dirs, sizeof(*dirs) * window_size);
    CUDAASSERT(err);
    err = hipMalloc((void **) &gpu_costs, sizeof(*costs) * window_size);
    CUDAASSERT(err);
    err = hipMalloc((void **) &gpu_vals, sizeof(*vals) * window_size);
    CUDAASSERT(err);
    err = hipMalloc((void **) &gpu_inverted_image, sizeof(*inverted_image) * window_size * 4);
    CUDAASSERT(err);

    // copy in our current values
    hipMemcpy(gpu_vals, vals, sizeof(*vals) * window_size, hipMemcpyHostToDevice);
    hipMemcpy(gpu_inverted_image, inverted_image, sizeof(*inverted_image) * window_size * 4, hipMemcpyHostToDevice);

    cout << "Initialized Seam Carver!" << endl;
    cout << "window width: "    << window_width << endl;
    cout << "window height: "   << window_height << endl;
    cout << "window size : "    << window_size << endl;

    glutInit(&argc, argv);

    glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE | GLUT_DEPTH);
    glutInitWindowSize(window_width, window_height);
    glutCreateWindow("Seam Carving");

    glutDisplayFunc(display);

    glEnable(GL_DEPTH_TEST);
    glClearColor(0.0, 0.0, 0.0, 1.0);

    glutMainLoop();
}
